#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

extern "C"
{
#include "png_util.h"
}

// to compile

// to convert output png files to an mp4 movie:
// ffmpeg -y -start_number 0 -r 24 -i gol%05d.png -b:v 8192k -c:v mpeg4 gol.mp4

/* function to convert from (i,j) cell index to linear storage index */
int idx(int N, int i, int j){
  int n = i + (N + 2)*j;
  return n;  
}

/* function to update Inew from Iold */
__global__ void cudaIterate(int N, float *Iold, float *Inew){
  int t = threadIdx.x;
  int b = blockIdx.x;
  int d = blockDim.x;

  int n = d*b +t;
  if(n<N+1&& n>0){
    int len = N+2;
    int surroundings = Iold[n-1] + Iold[n+1] + Iold[n+len] + Iold[n+len-1] +
      Iold[n+len+1] + Iold[n-len-1] + Iold[n-len] + Iold[n-len+1];
    int oldState = Iold[n];
    int newState = (oldState==1)?((surroundings==2)||(surroundings==3)):(surroundings==3);
    Inew[n] = newState;
  }
}

/* function to print game board for debugging */
void print_board(int N, float *board){
  printf("\n");
  for(int i=1; i<N+1; i=i+1){
    for(int j=1; j<N+1; j=j+1){
      printf("%d", (int)board[idx(N,i,j)]);
    }
    printf("\n");
  }
  printf("\n");
}

/* function to solve for game board using Game of Life rules */
void solve(int N){

  
  /* Intializes integer random number generator */
  srand(123456);

  // notice the size of these arrays
  float* h_Inew = (float*) calloc((N+2)*(N+2),sizeof(float));
  float* h_Iold = (float*) calloc((N+2)*(N+2),sizeof(float));
  float *c_Inew, *c_Iold;
  hipMalloc(&c_Inew, (N+2)*(N+2)*sizeof(float));
  hipMalloc(&c_Iold, (N+2)*(N+2)*sizeof(float));
  
  for(int i=1;i<N+1;i=i+1){
    for(int j=1;j<N+1;j=j+1){
      // set board state randomly to 1 or 0 
      h_Iold[idx(N,i,j)] = rand()%2;
    ls}
  }

  hipMemcpy(c_Iold, h_Iold, (N+2)*(N+2)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(c_Inew, h_Inew, (N+2)*(N+2)*sizeof(float), hipMemcpyHostToDevice);
  /* iterate here */
  int B = 256;
  int G = (N+B-1)/B;
  int count = 0;   // step counter
  int iostep = 10; // output every iostep
  int output = 1;  // save images if output=1
  int maxsteps = 1000; // maximum number of steps
  do{
    /* iterate from Iold to Inew */
    cudaIterate<<<G, B>>>(N, c_Iold, c_Inew);
    
    /* iterate from Inew to Iold */
    cudaIterate<<<G, B>>>(N, c_Inew, c_Iold);
    hipMemcpy(h_Iold, c_Iold, (N+2)*(N+2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_Inew, c_Inew, (N+2)*(N+2) * sizeof(float), hipMemcpyDeviceToHost);
    if(output==1 && count%iostep==0){
      char filename[BUFSIZ];
      FILE *png;
      sprintf(filename, "gol%05d.png", count/iostep);
      png = fopen(filename, "w");
      write_gray_png(png, N+2, N+2, h_Iold, 0, 1);
      fclose(png);
    }
    
    /* update counter */
    count = count + 1;
  }while(memcmp(h_Inew, h_Iold, (N+2)*(N+2)*sizeof(int))!=0 && count <= maxsteps);
  
  /* print out the cell existence in the whole board, then in cell (1 1) and (10 10)*/
  printf("I_{1 1} = %d\n",   (int)h_Iold[idx(N,1,1)]);
  printf("I_{10 10} = %d\n", (int)h_Iold[idx(N,10,10)]);
  printf("Took %d steps\n", count);
  free(h_Inew);
  free(h_Iold);
}



/* usage: ./main 100 
         to iterate, solve and display the game board of size N*/
int main(int argc, char **argv){

  if(argc!=2){
    printf("To run with an (N+2)x(N+2) board: \n");
    printf("  ./gameOfLife N\n");
    exit(-1);
  }
  
  /* start timer */
  clock_t begin = clock();
  /* read N from the command line arguments */
  int N = atoi(argv[1]);

  /* to solve for cell existence in game of life game board */
  solve(N);

  /* end timer*/
  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  printf("Time spent = %g sec\n", time_spent);
  return 0;
}
